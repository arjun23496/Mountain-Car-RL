#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void (int *a, int *b, int *c){
	*c = *a + *b;
}


int main(){
	int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);

	printf("%d\n", size);

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	a = 2;
	b = 3;

	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	add<<<1,1>>>(d_a, d_b, d_c);

	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	printf("%d\n", c );

	hipFree(d_a); hipFree(d_b); hipFree(d_c);
}